#include "VisibilityHist.h"

void VisibilityHistogram::Init(int screenWidth, int screenHeight, VolumeDataset &volume)
{
	HANDLE_ERROR( hipMalloc((void**)&cudaTexture, volume.xRes * volume.yRes * volume.zRes * volume.bytesPerElement) );
	HANDLE_ERROR( hipMemcpy(cudaTexture, volume.memblock3D, volume.xRes * volume.yRes * volume.zRes * volume.bytesPerElement, hipMemcpyHostToDevice) );

	xPixels = screenWidth;
	yPixels = screenHeight;

}



void VisibilityHistogram::CalculateHistogram(VolumeDataset &volume, TransferFunction &transferFunction, Camera &camera, ShaderManager shaderManager)
{



}