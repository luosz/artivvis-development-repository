#include "hip/hip_runtime.h"
#include "TempCoherence.h"


TempCoherence::TempCoherence(int screenWidth, int screenHeight, VolumeDataset &volume)
{
	blockRes = 8;


	numXBlocks = glm::ceil((float)volume.xRes / (float)blockRes);
	numYBlocks = glm::ceil((float)volume.yRes / (float)blockRes);
	numZBlocks = glm::ceil((float)volume.zRes / (float)blockRes);
	numBlocks = numXBlocks * numYBlocks * numZBlocks;

	float xVoxelWidth = 2.0f / (float) volume.xRes;
	float yVoxelWidth = 2.0f / (float) volume.yRes;
	float zVoxelWidth = 2.0f / (float) volume.zRes;
}

