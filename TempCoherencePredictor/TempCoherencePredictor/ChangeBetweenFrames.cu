#include "hip/hip_runtime.h"
#include "ChangeBetweenFrames.h"

void ChangeBetweenFrames::Init(VolumeDataset &volume)
{
	cudaResources.push_back(hipGraphicsResource_t());
	cudaResources.push_back(hipGraphicsResource_t());

	prevTexture3D = Generate3DTexture(volume);

	HANDLE_ERROR( hipMalloc((void**)&l1, sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&l2, sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&l3, sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&l4, sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&l5, sizeof(float)) );
}


GLuint ChangeBetweenFrames::Generate3DTexture(VolumeDataset &volume)
{
	GLuint tex;

	texture3DSize = volume.xRes * volume.yRes * volume.zRes * volume.bytesPerElement;

	glEnable(GL_TEXTURE_3D);
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_3D, tex);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_EDGE);

	glTexImage3D(GL_TEXTURE_3D, 0, GL_R8, volume.xRes, volume.yRes, volume.zRes, 0,  GL_RED, GL_UNSIGNED_BYTE, volume.memblock3D);

	glBindTexture(GL_TEXTURE_3D, 0);

	return tex;
}

texture <unsigned char, hipTextureType3D, hipReadModeElementType> prevTexRef;
texture <unsigned char, hipTextureType3D, hipReadModeElementType> currTexRef;

__global__ void CudaFindDiffBetweenFrames(int numVoxels, int xRes, int yRes, int zRes, float *l1, float *l2, float *l3, float *l4, float *l5)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);

	if (tid < numVoxels)
	{
		int z = tid / (xRes * yRes);
		int remainder = tid % (xRes * yRes);

		int y = remainder / xRes;

		int x = remainder % xRes;

		unsigned char prevVal, currVal;

		prevVal = tex3D(prevTexRef, x, y, z);
		currVal = tex3D(currTexRef, x, y, z);

		float percent = (float)glm::abs(prevVal - currVal);

		if (prevVal != (unsigned char)0)
			percent *= (100.0f / (float)prevVal);
		else
			percent *= 100.0f;

	//	printf ("%u - %u - %f\n", prevVal, currVal, percent);

		if (percent < 0.5f)
			atomicAdd(l1, (float)1.0f);
		else if (percent < 1.0f)
			atomicAdd(l2, (float)1.0f);
		else if (percent < 3.0f)
			atomicAdd(l3, (float)1.0f);
		else if (percent < 10.0f)
			atomicAdd(l4, (float)1.0f);
		else
			atomicAdd(l5, (float)1.0f);
	}
}

void ChangeBetweenFrames::Find(VolumeDataset &volume, int currentTimestep, GLuint bruteTex3D)
{
	HANDLE_ERROR( hipMemset(l1, 0, sizeof(float)) );
	HANDLE_ERROR( hipMemset(l2, 0, sizeof(float)) );
	HANDLE_ERROR( hipMemset(l3, 0, sizeof(float)) );
	HANDLE_ERROR( hipMemset(l4, 0, sizeof(float)) );
	HANDLE_ERROR( hipMemset(l5, 0, sizeof(float)) );


	glBindTexture(GL_TEXTURE_3D, prevTexture3D);
	glTexImage3D(GL_TEXTURE_3D, 0, GL_R8, volume.xRes, volume.yRes, volume.zRes, 0,  GL_RED, GL_UNSIGNED_BYTE, (volume.memblock3D + (texture3DSize * (currentTimestep-1))));
	glBindTexture(GL_TEXTURE_3D, 0);

	HANDLE_ERROR( hipGraphicsGLRegisterImage(&cudaResources[0], prevTexture3D, GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone) );
	HANDLE_ERROR( hipGraphicsGLRegisterImage(&cudaResources[1], bruteTex3D, GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone) );
	HANDLE_ERROR( hipGraphicsMapResources(2, &cudaResources[0]) );

	hipArray *prevArry = 0;	
	HANDLE_ERROR( hipGraphicsSubResourceGetMappedArray(&prevArry, cudaResources[0], 0, 0) ); 
	HANDLE_ERROR( hipBindTextureToArray(prevTexRef, prevArry) );

	hipArray *currArry = 0;	
	HANDLE_ERROR( hipGraphicsSubResourceGetMappedArray(&currArry, cudaResources[1], 0, 0) ); 
	HANDLE_ERROR( hipBindTextureToArray(currTexRef, currArry) );

	CudaFindDiffBetweenFrames <<<(volume.numVoxels + 255) / 256, 256>>> (volume.numVoxels, volume.xRes, volume.yRes, volume.zRes, l1, l2, l3, l4, l5);

	HANDLE_ERROR( hipUnbindTexture(prevTexRef) );
	HANDLE_ERROR( hipUnbindTexture(currTexRef) );

	HANDLE_ERROR( hipGraphicsUnmapResources(2, &cudaResources[0]) );
	HANDLE_ERROR( hipGraphicsUnregisterResource(cudaResources[0]) );
	HANDLE_ERROR( hipGraphicsUnregisterResource(cudaResources[1]) );

	HANDLE_ERROR( hipMemcpy(&la1, l1, sizeof(float), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(&la2, l2, sizeof(float), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(&la3, l3, sizeof(float), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(&la4, l4, sizeof(float), hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(&la5, l5, sizeof(float), hipMemcpyDeviceToHost) );

	la1 /= volume.numVoxels;
	la2 /= volume.numVoxels;
	la3 /= volume.numVoxels;
	la4 /= volume.numVoxels;
	la5 /= volume.numVoxels;

//	std::cout << la1 << " - " << la2 << " - " << la3 << " - " << la4 << " - " << la5 << std::endl;
}
