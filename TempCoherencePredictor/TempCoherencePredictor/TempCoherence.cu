#include "hip/hip_runtime.h"
#include "TempCoherence.h"

texture <unsigned char, hipTextureType3D, hipReadModeElementType> prevTexRef;
texture <unsigned char, hipTextureType3D, hipReadModeElementType> currTexRef;
texture <unsigned char, hipTextureType3D, hipReadModeElementType> nextTexRef;

TempCoherence::TempCoherence(VolumeDataset &volume)
{
	epsilon = 10;
	blockRes = 8;
	alpha = 6;

	numXBlocks = glm::ceil((float)volume.xRes / (float)blockRes);
	numYBlocks = glm::ceil((float)volume.yRes / (float)blockRes);
	numZBlocks = glm::ceil((float)volume.zRes / (float)blockRes);
	numBlocks = numXBlocks * numYBlocks * numZBlocks;

	float xVoxelWidth = 2.0f / (float) volume.xRes;
	float yVoxelWidth = 2.0f / (float) volume.yRes;
	float zVoxelWidth = 2.0f / (float) volume.zRes;

	currentTimestep = 0;

	textureSize = volume.xRes * volume.yRes * volume.zRes * volume.bytesPerElement;
	prevTexture3D = GenerateTexture(volume);
	currTexture3D = GenerateTexture(volume);
	nextTexture3D = GenerateTexture(volume);

	for (int i=0; i<3; i++)
		cudaResources.push_back(hipGraphicsResource_t());

	prevTempVolume = new unsigned char[volume.numVoxels * volume.bytesPerElement];
	currTempVolume = new unsigned char[volume.numVoxels * volume.bytesPerElement];
	nextTempVolume = new unsigned char[volume.numVoxels * volume.bytesPerElement];
	chunkToBeCopied = new unsigned char[numBlocks * blockRes * blockRes * blockRes];

	HANDLE_ERROR( hipMalloc((void**)&cudaCopiedChunk, numBlocks * blockRes * blockRes * blockRes) );

	blocksToBeCopied.resize(numBlocks);

	frequencyHistogram.resize(256);

	ratioTimeSteps = 200;
	ratios.resize(ratioTimeSteps);
	std::fill(ratios.begin(), ratios.end(), 0.0f);
}

__global__ void CudaPredict(int numVoxels, int xRes, int yRes, int zRes, hipSurfaceObject_t surface)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);

	if (tid < numVoxels)
	{
		int z = tid / (xRes * yRes);
		int remainder = tid % (xRes * yRes);

		int y = remainder / xRes;

		int x = remainder % xRes;

		unsigned char prevVal, currVal, nextVal;

		prevVal = tex3D(prevTexRef, x, y, z);
		currVal = tex3D(currTexRef, x, y, z);

		int temp = (EXTRAP_CONST * currVal) - prevVal;
		nextVal = (unsigned char)glm::clamp(temp, 0, 255);

		surf3Dwrite(nextVal, surface, x, y, z);
	}
}

void TempCoherence::GPUPredict(VolumeDataset &volume)
{
	HANDLE_ERROR( hipGraphicsGLRegisterImage(&cudaResources[0], prevTexture3D, GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone) );
	HANDLE_ERROR( hipGraphicsGLRegisterImage(&cudaResources[1], currTexture3D, GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone) );
	HANDLE_ERROR( hipGraphicsGLRegisterImage(&cudaResources[2], nextTexture3D, GL_TEXTURE_3D, hipGraphicsRegisterFlagsSurfaceLoadStore) );

	HANDLE_ERROR( hipGraphicsMapResources(3, &cudaResources[0]) );

	hipArray *prevArry = 0;	
	HANDLE_ERROR( hipGraphicsSubResourceGetMappedArray(&prevArry, cudaResources[0], 0, 0) ); 
	HANDLE_ERROR( hipBindTextureToArray(prevTexRef, prevArry) );

	hipArray *currArry = 0;	
	HANDLE_ERROR( hipGraphicsSubResourceGetMappedArray(&currArry, cudaResources[1], 0, 0) ); 
	HANDLE_ERROR( hipBindTextureToArray(currTexRef, currArry) );

	hipArray *nextArry = 0;	
	HANDLE_ERROR( hipGraphicsSubResourceGetMappedArray(&nextArry, cudaResources[2], 0, 0) ); 

	hipResourceDesc wdsc;
	wdsc.resType = hipResourceTypeArray;
	wdsc.res.array.array = nextArry;
	hipSurfaceObject_t writeSurface;
	HANDLE_ERROR( hipCreateSurfaceObject(&writeSurface, &wdsc) );

	CudaPredict <<<(volume.numVoxels + 255) / 256, 256>>>(volume.numVoxels, volume.xRes, volume.yRes, volume.zRes, writeSurface);

	// Unbind and unmap, must be done before OpenGL uses texture memory again
	HANDLE_ERROR( hipUnbindTexture(prevTexRef) );
	HANDLE_ERROR( hipUnbindTexture(currTexRef) );
	HANDLE_ERROR( hipUnbindTexture(nextTexRef) );

	HANDLE_ERROR( hipGraphicsUnmapResources(3, &cudaResources[0]) );

	HANDLE_ERROR( hipGraphicsUnregisterResource(cudaResources[0]) );
	HANDLE_ERROR( hipGraphicsUnregisterResource(cudaResources[1]) );
	HANDLE_ERROR( hipGraphicsUnregisterResource(cudaResources[2]) );

}

/*
bool TempCoherence::BlockCompare(VolumeDataset &volume, int x, int y, int z)
{
	GLubyte *nextVolume = volume.memblock3D + (currentTimestep * volume.numVoxels);

	int xMin = x * blockRes;
	int yMin = y * blockRes;
	int zMin = z * blockRes;

	int ID;
	float omega, beta;
	float top, bottom;
	top = bottom = 0.0f;

	for (int k=0; k<blockRes; k++)
		for (int j=0; j<blockRes; j++)
			for (int i=0; i<blockRes; i++)
			{
				if ((xMin + i) >= volume.xRes || (yMin + j) >= volume.yRes || (zMin + k) >= volume.zRes)
					continue;

				ID = (xMin + i) + ((yMin + j) * volume.xRes) + ((zMin + k) * volume.xRes * volume.yRes);

				unsigned char p = nextTempVolume[ID];
				unsigned char n = nextVolume[ID];

				if (n <= alpha)
					beta = (float)n / float(alpha);
				else
					beta = ((float)(255 - n)) / ((float)(255 - alpha));

				omega = ((float)frequencyHistogram[n] / (float) maxFrequency);

//				omega = beta;

				int diff =  n - p;
				
				top += omega * glm::pow(diff, 2);

//				bottom += omega;
			}

//	bottom *= nonZeroFrequencies;
	bottom = blockRes * blockRes * blockRes;

	float similar = glm::sqrt(top / bottom);
//	similar = glm::sqrt(top);

	if (similar < (float)epsilon)
		return true;


	for (int k=0; k<blockRes; k++)
		for (int j=0; j<blockRes; j++)
			for (int i=0; i<blockRes; i++)
			{
				if ((xMin + i) >= volume.xRes || (yMin + j) >= volume.yRes || (zMin + k) >= volume.zRes)
					continue;

				ID = (xMin + i) + ((yMin + j) * volume.xRes) + ((zMin + k) * volume.xRes * volume.yRes);

				currTempVolume[ID] = nextVolume[ID];
			}

	return false;
}
*/


void TempCoherence::CopyBlockToGPU(VolumeDataset &volume, hipArray *nextArry, int x, int y, int z)
{
	GLubyte *currentTimeAddress = volume.memblock3D + (currentTimestep * volume.numVoxels);
	hipPos offset = make_hipPos((x * blockRes), (y * blockRes), (z * blockRes));
	hipExtent extent = make_hipExtent(blockRes, blockRes, blockRes);

	hipMemcpy3DParms cudaCpyParams = {0};
	cudaCpyParams.kind = hipMemcpyHostToDevice;
	cudaCpyParams.extent = extent;

	cudaCpyParams.dstPos = offset;
	cudaCpyParams.dstArray = nextArry;
	
	cudaCpyParams.srcPos = offset;
	cudaCpyParams.srcPtr = make_hipPitchedPtr((void*)currentTimeAddress, volume.xRes, volume.yRes, volume.zRes);

	hipMemcpy3D(&cudaCpyParams);
}

void TempCoherence::CopyBlockToChunk(VolumeDataset &volume, int x, int y, int z)
{
	GLubyte *currentTimeAddress = volume.memblock3D + (currentTimestep * volume.numVoxels);
	hipExtent extent = make_hipExtent(blockRes, blockRes, blockRes);

//	if (x == numXBlocks - 1)
//		extent.width = volume.xRes % blockRes;
//	if (y == numYBlocks - 1)
//		extent.height = volume.yRes % blockRes;
//	if (z == numZBlocks - 1)
//		extent.depth = volume.zRes % blockRes;

	hipMemcpy3DParms cudaCpyParams = {0};
	cudaCpyParams.kind = hipMemcpyHostToHost;
	cudaCpyParams.extent = extent;

	cudaCpyParams.srcPos = make_hipPos((x * blockRes), (y * blockRes), (z * blockRes));
	cudaCpyParams.srcPtr = make_hipPitchedPtr((void*)currentTimeAddress, volume.xRes, volume.yRes, volume.zRes);

	cudaCpyParams.dstPos = make_hipPos((numBlocksCopied * blockRes), 0, 0);
	cudaCpyParams.dstPtr = make_hipPitchedPtr((void*)chunkToBeCopied, numBlocks * blockRes, blockRes, blockRes);
	
	hipMemcpy3D(&cudaCpyParams) ;
}

void TempCoherence::CPUPredict(VolumeDataset &volume)
{
	std::fill(frequencyHistogram.begin(), frequencyHistogram.end(), 0);

	// Beware of this, think it requires even stepsize
	for (int i=0; i<volume.numVoxels; i++)
	{
		int temp = (EXTRAP_CONST * currTempVolume[i]) - prevTempVolume[i];
		nextTempVolume[i] = (unsigned char)glm::clamp(temp, 0, 255);

		prevTempVolume[i] = currTempVolume[i];
		currTempVolume[i] = nextTempVolume[i];	
	}

	for (int i=0; i<volume.numVoxels; i++)
	{
		int bucket = volume.memblock3D[(currentTimestep*volume.numVoxels) + i];
		frequencyHistogram[bucket]++;
	}

	maxFrequency = nonZeroFrequencies = 0;
	for (int i=1; i<256; i++)
	{
		int freq = frequencyHistogram[i];
		maxFrequency = glm::max(maxFrequency, freq);
		nonZeroFrequencies += freq;
	}
	frequencyHistogram[0] = maxFrequency;

	for (int z=0; z<numZBlocks; z++)
		for (int y =0; y<numYBlocks; y++)
			for (int x=0; x<numXBlocks; x++)
			{
				if (BlockCompare(volume, x, y, z) == false)
				{
					blocksToBeCopied[numBlocksCopied] = BlockID(x, y, z);
					CopyBlockToChunk(volume, x, y, z);

					numBlocksCopied++;
				}
				else
					numBlocksExtrapolated++;
			} 
}


void TempCoherence::CopyChunkToGPU(VolumeDataset &volume)
{
	hipExtent extent = make_hipExtent(numBlocksCopied * blockRes, blockRes, blockRes);

	hipMemcpy3DParms cudaCpyParams = {0};
	cudaCpyParams.kind = hipMemcpyHostToDevice;
	cudaCpyParams.extent = extent;

	cudaCpyParams.srcPtr = make_hipPitchedPtr((void*)chunkToBeCopied, numBlocks * blockRes, blockRes, blockRes);

	cudaCpyParams.dstPtr = make_hipPitchedPtr((void*)cudaCopiedChunk, numBlocks * blockRes, blockRes, blockRes);
	
	hipMemcpy3D(&cudaCpyParams);


	HANDLE_ERROR( hipGraphicsGLRegisterImage(&cudaResources[0], nextTexture3D, GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone) );
	HANDLE_ERROR( hipGraphicsMapResources(1, &cudaResources[0]) );
	hipArray *nextArry = 0;	
	HANDLE_ERROR( hipGraphicsSubResourceGetMappedArray(&nextArry, cudaResources[0], 0, 0) ); 
	HANDLE_ERROR( hipBindTextureToArray(nextTexRef, nextArry) );


	extent = make_hipExtent(blockRes, blockRes, blockRes);

	cudaCpyParams = hipMemcpy3DParms();
	cudaCpyParams.kind = hipMemcpyDeviceToDevice;
	cudaCpyParams.extent = extent;

	for (int i=0; i<numBlocksCopied; i++)
	{
		cudaCpyParams.srcPos = make_hipPos((i * blockRes), 0, 0);
		cudaCpyParams.srcPtr = make_hipPitchedPtr((void*)cudaCopiedChunk, numBlocks * blockRes, blockRes, blockRes);
		
		cudaCpyParams.dstPos = make_hipPos((blocksToBeCopied[i].x * blockRes), (blocksToBeCopied[i].y * blockRes), (blocksToBeCopied[i].z * blockRes));
		cudaCpyParams.dstArray = nextArry;

		hipMemcpy3D(&cudaCpyParams) ;
	}

	HANDLE_ERROR( hipUnbindTexture(nextTexRef) );
	HANDLE_ERROR( hipGraphicsUnmapResources(1, &cudaResources[0]) );
	HANDLE_ERROR( hipGraphicsUnregisterResource(cudaResources[0]) );
}


GLuint TempCoherence::TemporalCoherence(VolumeDataset &volume, int currentTimestep_)
{
	currentTimestep = currentTimestep_;
	numBlocksCopied = numBlocksExtrapolated = 0;

	GLuint temp = prevTexture3D;
	prevTexture3D = currTexture3D;
	currTexture3D = nextTexture3D;
	nextTexture3D = temp;

	if (currentTimestep < 2)
	{
		glBindTexture(GL_TEXTURE_3D, nextTexture3D);
		glTexImage3D(GL_TEXTURE_3D, 0, GL_R8, volume.xRes, volume.yRes, volume.zRes, 0,  GL_RED, GL_UNSIGNED_BYTE, (volume.memblock3D + (textureSize * currentTimestep)));
		glBindTexture(GL_TEXTURE_3D, 0);

		if (currentTimestep == 1)
		{
			for (int i=0; i<volume.numVoxels; i++)
			{
				prevTempVolume[i] = volume.memblock3D[i];
				currTempVolume[i] = volume.memblock3D[textureSize + i];
			}						
		}
	}
	else
	{
//		if (currentTimestep == ratioTimeSteps)
//		{
//			maxRatio = 0.0f;
//			minRatio = 100.0f;
//			meanRatio = 0.0f;
//			stdDev = 0.0f;
//
//			for (int i=2; i<ratioTimeSteps; i++)
//			{
//				maxRatio = glm::max(maxRatio, ratios[i]);
//				minRatio = glm::min(minRatio, ratios[i]);
//				meanRatio += ratios[i];
//			}
//
//			meanRatio /= ratioTimeSteps;
//
//			for (int i=2; i<ratioTimeSteps; i++)
//			{
//				stdDev += glm::pow((ratios[i] - meanRatio), 2.0f);
//			}
//
//			stdDev /= ratioTimeSteps;
//			stdDev = glm::sqrt(stdDev);
//
//			std::cout << "Max: " << maxRatio << std::endl;
//			std::cout << "Min: " << minRatio << std::endl;
//			std::cout << "Mean: " << meanRatio << std::endl;
//			std::cout << "StdDev: " << stdDev << std::endl;
////			getchar();
//		}

		GPUPredict(volume);
		CPUPredict(volume);

		CopyChunkToGPU(volume);


//		glBindTexture(GL_TEXTURE_3D, nextTexture3D);
//		glTexImage3D(GL_TEXTURE_3D, 0, GL_R8, volume.xRes, volume.yRes, volume.zRes, 0,  GL_RED, GL_UNSIGNED_BYTE, (volume.memblock3D + (textureSize * currentTimestep)));
//		glBindTexture(GL_TEXTURE_3D, 0);

	}

//	std::cout << "Copied: " << numBlocksCopied << " - Extrapolated: " << numBlocksExtrapolated << std::endl;
//	ratios[currentTimestep] = (float)numBlocksExtrapolated / (float) numBlocks;
	
	return nextTexture3D;
}


GLuint TempCoherence::GenerateTexture(VolumeDataset &volume)
{
	GLuint tex;

	glEnable(GL_TEXTURE_3D);
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_3D, tex);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_EDGE);

	glTexImage3D(GL_TEXTURE_3D, 0, GL_R8, volume.xRes, volume.yRes, volume.zRes, 0,  GL_RED, GL_UNSIGNED_BYTE, (volume.memblock3D + (textureSize * currentTimestep)));

	glBindTexture(GL_TEXTURE_3D, 0);

	return tex;
}





bool TempCoherence::BlockCompare(VolumeDataset &volume, int x, int y, int z)
{
	GLubyte *nextVolume = volume.memblock3D + (currentTimestep * volume.numVoxels);

	int xMin = x * blockRes;
	int yMin = y * blockRes;
	int zMin = z * blockRes;

	int ID;

	for (int k=0; k<blockRes; k++)
		for (int j=0; j<blockRes; j++)
			for (int i=0; i<blockRes; i++)
			{
				if ((xMin + i) >= volume.xRes || (yMin + j) >= volume.yRes || (zMin + k) >= volume.zRes)
					continue;

				ID = (xMin + i) + ((yMin + j) * volume.xRes) + ((zMin + k) * volume.xRes * volume.yRes);

				unsigned char p = nextTempVolume[ID];
				unsigned char n = nextVolume[ID];

				int diff =  p - n;
				int absDiff = glm::abs(diff);

				if (absDiff > epsilon)
					goto copy;
			}

	return true;

	// Put a goto to avoid an extra if, only gets here if entire block needs to be copied
	copy:
	for (int k=0; k<blockRes; k++)
		for (int j=0; j<blockRes; j++)
			for (int i=0; i<blockRes; i++)
			{
				if ((xMin + i) >= volume.xRes || (yMin + j) >= volume.yRes || (zMin + k) >= volume.zRes)
					continue;

				ID = (xMin + i) + ((yMin + j) * volume.xRes) + ((zMin + k) * volume.xRes * volume.yRes);

				currTempVolume[ID] = nextVolume[ID];
			}

	return false;
}
