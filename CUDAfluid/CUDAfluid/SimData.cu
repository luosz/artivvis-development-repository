#include "SimData.h"


void SimData::Init()
{
	// Allocate Memory
	HANDLE_ERROR( hipMalloc((void**)&gridRes, 3 * sizeof(int)) );
	HANDLE_ERROR( hipMalloc((void**)&numCells,  sizeof(int)) );

	HANDLE_ERROR( hipMalloc((void**)&xFaceRes, 3 * sizeof(int)) );
	HANDLE_ERROR( hipMalloc((void**)&yFaceRes, 3 * sizeof(int)) );
	HANDLE_ERROR( hipMalloc((void**)&zFaceRes, 3 * sizeof(int)) );

	HANDLE_ERROR( hipMalloc((void**)&numFaces, 3 * sizeof(int)) );

	HANDLE_ERROR( hipMalloc((void**)&dt,  sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&dx,  sizeof(float)) );

	HANDLE_ERROR( hipMalloc((void**)&xVelocities, (gridXRes+1) * gridYRes * gridZRes * sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&yVelocities, gridXRes * (gridYRes+1) * gridZRes * sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&zVelocities, gridXRes * gridYRes * (gridZRes+1) * sizeof(float)) );

	HANDLE_ERROR( hipMalloc((void**)&pressures, numGridCells * sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&densities, numGridCells * sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void**)&temperatures, numGridCells * sizeof(float)) );

	int Xp1 = gridXRes + 1;
	int Yp1 = gridYRes + 1;
	int Zp1 = gridZRes + 1;

	// Initialize values
	HANDLE_ERROR( hipMemcpy(&gridRes[0], &gridXRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&gridRes[1], &gridYRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&gridRes[2], &gridZRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(numCells, &numGridCells, sizeof(int), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(&xFaceRes[0], &Xp1, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&xFaceRes[1], &gridYRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&xFaceRes[2], &gridZRes, sizeof(int), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(&yFaceRes[0], &gridXRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&yFaceRes[1], &Yp1, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&yFaceRes[2], &gridZRes, sizeof(int), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(&zFaceRes[0], &gridXRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&zFaceRes[1], &gridYRes, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&zFaceRes[2], &Zp1, sizeof(int), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(&numFaces[0], &numXFaces, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&numFaces[1], &numYFaces, sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(&numFaces[2], &numZFaces, sizeof(int), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(dt, &timestep, sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(dx, &cellSize, sizeof(float), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemset(xVelocities, 0, numXFaces * sizeof(float)) );
	HANDLE_ERROR( hipMemset(yVelocities, 0, numYFaces * sizeof(float)) );
	HANDLE_ERROR( hipMemset(zVelocities, 0, numZFaces * sizeof(float)) );

	HANDLE_ERROR( hipMemset(pressures, 0, numGridCells * sizeof(float)) );
	HANDLE_ERROR( hipMemset(densities, 0, numGridCells * sizeof(float)) );
	HANDLE_ERROR( hipMemset(temperatures, 0, numGridCells * sizeof(float)) );

	hipDeviceSynchronize();
}